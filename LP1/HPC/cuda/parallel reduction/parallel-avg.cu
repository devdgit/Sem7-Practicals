#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include<bits/stdc++.h>
#include <iostream>
#include <numeric>

using namespace std;

#define BLOCK_SIZE 4

__global__ void avg(int* input)				// kernel function definition
{
	const int tid = threadIdx.x;
	int avg=0;
	int index=0;
	int step_size = 1;
	int number_of_threads = blockDim.x;		// blockDim = 4 i.e. number of threads per block = 4
	

	while (number_of_threads > 0)
	{
		if (tid < number_of_threads) // still alive?
		{     
			const int fst = tid * step_size * 2;		//get the index in array
			const int snd = fst + step_size;			//get the index in array
			input[fst] += input[snd];
			avg =input[fst];
			index=fst;
			
			//input[fst]=input[fst]/2;
			
		}

		step_size <<= 1; 			// increment step_size by 1
		number_of_threads >>= 1;	//decrement number of threads by 2
	}
	input[index]=avg/7;				// calculate average
}

int main()
{
	const int count = 8 ;
	
	
	const int size = count * sizeof(int);
	
	
	int h[] = {13, 27, 15, 14, 33, 2, 24};

	int* d;						//GPU parameter
       	
	hipMalloc(&d, size);		//assign memory to parameters on GPU
	
	hipMemcpy(d, h, size, hipMemcpyHostToDevice);		//copy the array from CPU to GPU
	avg <<<1, count / 2 >>>(d);							// call kernel function <<<number of blocks, number of threads= number of elements/2
	
	int result;
	
	hipMemcpy(&result, d, sizeof(int), hipMemcpyDeviceToHost);		// copy the result back from GPU to CPU

	cout << "Avg is " << result << endl;

	getchar();

	hipFree(d);		// Free the allocated memory
	delete[] h;

	return 0;
}
