
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h> // for rand()
#include <errno.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_math_constants.h>

#define N 2*2*2

/* host buffer */
float *data;
/* device buffers */
float *dSrc, *dDst;

void check_error(hipError_t error, const char *message) {
    if (error != hipSuccess) {
        fprintf(stderr, "%s (%s)\n", message,
            hipGetErrorString(error));
        if (dSrc)
            hipFree(dSrc);
        if (dDst)
            hipFree(dDst);
        exit(1);
    }
}

#define WARP_SIZE 32
#define BLOCK_SIZE (12*WARP_SIZE)

__global__ void findmax(float *dDst, const float *dSrc, uint dim)
{
    __shared__ float cache[BLOCK_SIZE];					//cache is shared array between the block of threads

    uint gix = threadIdx.x + blockDim.x*blockIdx.x;

#define tid threadIdx.x

    float acc = HIP_NAN_F;		//acc is initialized to certain random floating point value

    while (gix < dim) {				//this code executed by each thread
        acc = fmax(acc, dSrc[gix]);
        gix += blockDim.x*gridDim.x;
    }

    cache[tid] = acc;				//each thread keeps its result in cache index

    uint active = blockDim.x >> 1;	//active is incremented by 1

    do {
        __syncthreads();			// guarantee that all threads are in the same iteration of the do- while loop at the same time
        if (tid < active)
            cache[tid] = fmax(cache[tid], cache[tid+active]);			//calculate final result by comparing the values kept by each thread in cache
        active >>= 1;
    } while (active > 0);

    if (tid == 0)					//executed by parent thread
        dDst[blockIdx.x] = cache[0];
}

int main(int argc, char **argv) {
    data = (float*) calloc(N, sizeof(float));
    size_t data_size = N * sizeof(float);				// Here, N = 8
    float max = nan(""), d_max = nan("");				// returns a value of type double

    for (size_t i = 0; i < N; ++i) {
        data[i] = i; 									// input array
        max = fmax(max, data[i]);						// calculate max value sequentially
    }
    printf("%u elements generated, max %g, data size %zu (%zuMB)\n",
            N, max, data_size, data_size>>20);

    hipError_t err;

    err = hipMalloc(&dSrc, data_size);
    check_error(err, "allocating array");

    err = hipMemcpy(dSrc, data, data_size, hipMemcpyHostToDevice);		// copy array to device
    check_error(err, "copy UP");

    uint nblocks = 8;

    err = hipMalloc(&dDst, nblocks*sizeof(*dDst));
    check_error(err, "allocating Dst array");

    hipEvent_t start, stop;
    float runtime;
    hipEventCreate(&start);				//Creates an event object for the current device
    hipEventCreate(&stop);					//Creates an event object for the current device

    hipEventRecord(start, 0);				// records an event. first parameter: event object, second parameter:stream in which to record the event, if value is zero, records the event after all preceding operations are completed
    findmax<<<nblocks,BLOCK_SIZE>>>(dDst, dSrc, N);		//call kernel function
    findmax<<<1,BLOCK_SIZE>>>(dDst, dDst, nblocks);
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);				//Wait until the completion of all device work preceding the most recent call to cudaEventRecord()
    hipEventElapsedTime(&runtime, start, stop);		//Computes the elapsed time between two events and stores in first parameter

    /* Giga-elements per second */
    printf("%u elements processed in %gms: %gGE/s\n",
        N, runtime, (N/runtime)/(1000000));

    /* Actual bandwith in GB/s */
    uint total_els = N + nblocks;
    float sizeMB = float(total_els)*sizeof(float)/(1024*1024);
    printf("Bandwidth: %u elements (%gMB) read in two steps. "
        "Runtime: %gms (%gGB/s)\n",
        total_els, sizeMB, runtime, sizeMB/runtime);

    err = hipMemcpy(&d_max, dDst, sizeof(d_max), hipMemcpyDeviceToHost);		//copy the result back to host
    check_error(err, "copy DOWN");

    hipFree(dSrc); dSrc = NULL;
    hipFree(dDst); dDst = NULL;
    free(data);

    printf("Parallel max: %g vs %g\n", d_max, max);
}

