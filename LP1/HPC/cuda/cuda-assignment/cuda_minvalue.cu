    #include <hip/hip_runtime.h>
    #include <stdio.h>
    #include <time.h>

    #define SIZE 10

    __global__ void max(int *a , int *c)	// kernel function definition
    {
    int i = threadIdx.x;					// initialize i to thread ID

    *c = a[0];

            if(a[i] < *c)
                    {
                    *c = a[i];
                    }

    }

    int main()
    {
    int i;
    srand(time(NULL));		//makes use of the computer's internal clock to control the choice of the seed

    int a[SIZE];
    int c;

    int *dev_a, *dev_c;		//GPU / device parameters

    hipMalloc((void **) &dev_a, SIZE*sizeof(int));		//assign memory to parameters on GPU from CUDA runtime API
    hipMalloc((void **) &dev_c, SIZE*sizeof(int));

    printf( "Enter the elements:\n");
  
	
     for(int i=0;i<SIZE;i++)
	{
		printf( "\n");
		scanf("%d",a[i]);
	}
	

    
    for( i = 0 ; i < SIZE ; i++)
    {
    	printf("%d", a[i]);			// input the numbers
    }
    
    hipMemcpy(dev_a , a, SIZE*sizeof(int),hipMemcpyHostToDevice);		//copy the array from CPU to GPU
    max<<<1,SIZE>>>(dev_a,dev_c);										// call kernel function <<<number of blocks, number of threads
    hipMemcpy(&c, dev_c, SIZE*sizeof(int),hipMemcpyDeviceToHost);		// copy the result back from GPU to CPU

    printf("\nmin =  %d ",c);

    hipFree(dev_a);		// Free the allocated memory
    hipFree(dev_c);
    printf("");

    return 0;
    }
