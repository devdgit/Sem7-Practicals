#include <hip/hip_runtime.h>
    #include <stdio.h>
    #include <time.h>

   // #define SIZE 1000
#define SIZE 10

    __global__ void max(int *a , int *c)	// kernel function definition
    {
    int i = threadIdx.x;					// initialize i to thread ID

    *c = a[0];

            if(a[i] < *c)
                    {
                    *c = a[i];
                    }

    }

    int main()
    {
    int i;
    srand(time(NULL));		//makes use of the computer's internal clock to control the choice of the seed

    int a[SIZE]={12,4,7,3,9,5,11,6,1,76};
    int c;

    int *dev_a, *dev_c;		//GPU / device parameters

    hipMalloc((void **) &dev_a, SIZE*sizeof(int));		//assign memory to parameters on GPU from CUDA runtime API
    hipMalloc((void **) &dev_c, SIZE*sizeof(int));

   // for( i = 0 ; i < SIZE ; i++)
    //{
    	//a[i] = i;			// input the numbers
   // }
    
    hipMemcpy(dev_a , a, SIZE*sizeof(int),hipMemcpyHostToDevice);		//copy the array from CPU to GPU
    max<<<1,SIZE>>>(dev_a,dev_c);										// call kernel function <<<number of blocks, number of threads
    hipMemcpy(&c, dev_c, SIZE*sizeof(int),hipMemcpyDeviceToHost);		// copy the result back from GPU to CPU

    printf("\nmin =  %d ",c);

    hipFree(dev_a);		// Free the allocated memory
    hipFree(dev_c);
    printf("");

    return 0;
    }                                                  
