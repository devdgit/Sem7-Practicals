
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
 
// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd(int *a, int *c, int n)
{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;
   // c[id]=0;
    // Make sure we do not go out of bounds
    if (id < n)
        *c+= a[id];
   // printf("\n%d", c[id]);
}
 
int main( int argc, char* argv[] )
{
    // Size of vectors
    // int n = 100000;
	int n=5;
 const int size = n * sizeof(int);  
    // Host input vectors
    int *h_a;
   // double *h_b;
    //Host output vector
    int *h_c;
 
    // Device input vectors
    int *d_a;
    //double *d_b;
    //Device output vector
    int *d_c;
    int dev=0;
    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(double);
 
    // Allocate memory for each vector on host
    //h_a = (int*)malloc(bytes);
    //h_b = (double*)malloc(bytes);
    h_c = (int*)malloc(bytes);
 
    // Allocate memory for each vector on GPU
    hipMalloc(&d_a, bytes);
   // cudaMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
 
    int i;
    printf("Input array");
    // Initialize vectors on host
    /*for( i = 0; i < n; i++ ) {
        // h_a[i] = sin(i)*sin(i);
        //printf("\n",i); 
	h_a[i]=i;
	//printf("\n%d", h_a[i]);
	//h_b[i]=i;
        //h_b[i] = cos(i)*cos(i);
    }*/
   
   int a[]= {0, 1, 2, 3, 4};
   
   hipMalloc(&h_a, size);
 
    // Copy host vectors to device
    hipMemcpy( h_a, a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( d_c, &dev, sizeof(int), hipMemcpyHostToDevice);
//    cudaMemcpy( d_b, h_b, bytes, cudaMemcpyHostToDevice);
 
    int blockSize, gridSize;
 
    // Number of threads in each thread block
    blockSize = 2;
 
    // Number of thread blocks in grid
    gridSize = (int)ceil((float)n/blockSize);
 
    // Execute the kernel
    vecAdd<<<gridSize, blockSize>>>(d_a,d_c,n);
      int result;
    // Copy array back to host
    hipMemcpy( &result,d_c, sizeof(int), hipMemcpyDeviceToHost );
 
    // Sum up vector c and print result divided by n, this should equal 1 within error
    double sum = 0;
    //for(i=0; i<n; i++)
      //  sum += h_c[i];
     
    printf("final result: %f\n",result );

   // vecdev<<<gridSize, blockSize>>>(d_a,d_c, n);
 
    // Release device memory
    hipFree(d_a);
    //cudaFree(d_b);
    hipFree(d_c);
 
    // Release host memory
    free(h_a);
    //free(h_b);
    free(h_c);
 
    return 0;
}
